
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 600

__global__ void VecAdd(int* DA, int* DB, int* DC)
{
	// int i = threadIdx.x;
  int i = threadIdx.x;
    DC[i] = DA[i] + DB[i];
}

int main()
{ int HA[N], HB[N], HC[N];
  int *DA, *DB, *DC;
  int i; int size = N*sizeof(int);
  
  // reservamos espacio en la memoria global del device
  hipError_t err_DA = hipHostMalloc((void**)&DA, size, hipHostMallocDefault);
  hipError_t err_DB = hipHostMalloc((void**)&DB, size, hipHostMallocDefault);
  hipError_t err_DC = hipHostMalloc((void**)&DC, size, hipHostMallocDefault);
  
  if (err_DA != hipSuccess) printf("%s\n", hipGetErrorString(err_DA));
  if (err_DB != hipSuccess) printf("%s\n", hipGetErrorString(err_DB));
  if (err_DC != hipSuccess) printf("%s\n", hipGetErrorString(err_DC));

  // inicializamos HA y HB
  for (i=0; i<N; i++) {HA[i]=-i; HB[i] = 3*i;}
  
  // copiamos HA y HB del host a DA y DB en el device, respectivamente
  hipError_t err_DAHA = hipMemcpy(DA, HA, size, hipMemcpyHostToDevice);
  hipError_t err_DBHB = hipMemcpy(DB, HB, size, hipMemcpyHostToDevice);
  
  if (err_DAHA != hipSuccess) printf("%s\n", hipGetErrorString(err_DAHA));
  if (err_DBHB != hipSuccess) printf("%s\n", hipGetErrorString(err_DBHB));

  // llamamos al kernel (1 bloque de N hilos)
  VecAdd <<<1, N>>>(DA, DB, DC);	// N hilos ejecutan el kernel en paralelo
  
  hipError_t errSync = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();

  if(errSync != hipSuccess) printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if(errAsync != hipSuccess) printf("Async kernel error: %s\n", hipGetErrorString(hipGetLastError()));

  // copiamos el resultado, que está en la memoria global del device, (DC) al host (a HC)
  hipError_t err_HCDC = hipMemcpy(HC, DC, size, hipMemcpyDeviceToHost);
  
  if (err_HCDC != hipSuccess) printf("%s\n", hipGetErrorString(err_HCDC));

  // liberamos la memoria reservada en el device
  hipHostFree(DA); hipHostFree(DB); hipHostFree(DC);
  
  // una vez que tenemos los resultados en el host, comprobamos que son correctos
  // esta comprobación debe quitarse una vez que el programa es correcto (p. ej., para medir el tiempo de ejecución)
  for (i = 0; i < N; i++) // printf("%d + %d = %d\n",HA[i],HB[i],HC[i]);
    if (HC[i]!= (HA[i]+HB[i])) 
		{printf("error en componente %d\n", i); break;}
    
  return 0;
} 
