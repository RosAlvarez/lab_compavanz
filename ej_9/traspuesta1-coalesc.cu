/* Copiar traspuesta de matriz h_a[F][C] en matriz h_b[C][F] aunque el n.º de hebras de 
   los bloques no divida al n.º de componentes de las matrices */
// #include <stdlib.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

#define F 25
#define C 43
// matriz original de F filas y C columnas
#define H 8
#define K 8
// bloques de H x K hebras (HxK<=512, cap. cpto. 1.3)

 __global__ void trspta1(int *dev_a, int *dev_b, int filas, int cols)
{ int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  
  if ((ix<cols)&&(iy<filas))
  { dev_b[ix*filas+iy] = dev_a[iy*cols+ix];}	// B[ix][iy] = A[iy][ix] (A matriz representada por dev_a, etc.)
}

__global__ void trspta_coal(int *dev_a, int *dev_b, int filas, int cols)
{
  __shared__ int tile[H][K];
 
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  
  if(ix < cols && iy < filas)
  {
    tile[threadIdx.y][threadIdx.x] = dev_a[iy*cols + ix];
  }

  ix = blockIdx.y * blockDim.x + threadIdx.x;
  iy = blockIdx.x * blockDim.y + threadIdx.y;

  if(ix < filas && iy < cols)
  {
    dev_b[iy*filas + ix] = tile[threadIdx.x][threadIdx.y];
  }
}

int main(int argc, char** argv)
{
  int h_a[F][C], h_b[C][F];
  int *d_a, *d_b;
  int i, j, aux, size = F * C * sizeof(int);
  dim3 hebrasBloque(K, H); // bloques de H x K hebras
  int numBlf = (F+H-1)/H;  // techo de F/H
  int numBlc = (C+K-1)/K;  // techo de C/K
  dim3 numBloques(numBlc,numBlf);

  // reservar espacio en el device para d_a y d_b
  hipMalloc((void**) &d_a, size); 
  hipMalloc((void**) &d_b, size);

  // dar valores a la matriz h_a en la CPU e imprimirlos
  printf("\nMatriz origen\n");
  for (i=0; i<F; i++) {
    for (j=0; j<C; j++) {
      aux = i*C+j;
      h_a[i][j] = aux;
      printf("%d ", aux);
    }
    printf("\n");
  }

  // copiar matriz h_a en d_a
  hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
  
  // llamar al kernel que obtiene en d_b la traspuesta de d_a
  trspta_coal<<<numBloques, hebrasBloque>>>(d_a, d_b, F, C);

  // copiar matriz d_b en h_b
  hipMemcpy(h_b, d_b, size, hipMemcpyDeviceToHost);
  
  // una vez que tenemos los resultados en el host, comprobamos que son correctos
  for (i=0; i<F; i++)
    for (j=0; j<C; j++) 
      if (h_a[i][j]!= h_b[j][i]) 
		{printf("error en componente %d %d de matriz de entrada \n", i,j); break;}
 
// imprimir matriz resultado
  printf("\nMatriz resultado\n");
  for (i=0; i<C; i++) {
    for (j=0; j<F; j++) {
      printf("%d ", h_b[i][j]);
    }
    printf("\n");
  }
  printf("\n");

  hipFree(d_a); hipFree(d_b);
  
  return 0;
} 
