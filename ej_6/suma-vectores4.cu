
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000000

__device__ int DA[N];
__device__  int DB[N];
__device__ int DC[N];
__device__ int stopKernel = 0;

__global__ void VecAdd()
{
	// int i = threadIdx.x;
  // for(int i=0; i<N; i++)
  // int i = blockIdx.x;
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N)  DC[i] = DA[i] + DB[i];
}

__global__ void initVec()
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    DA[i] = -i;
    DB[i] = 3*i;
}

__global__ void checkAdd()
{
    if(stopKernel == 0)
    {
      int i = blockIdx.x*blockDim.x + threadIdx.x;
      if (DC[i]!= (DA[i]+DB[i]))
      {
        printf("error en componente %d\n", i);
        stopKernel = 1;
      }
    }
}

int main()
{ 
  // iniciamos los valores de los vectores DA y DB
  initVec <<<(N+255)/256, 256>>>();

  // llamamos al kernel
  VecAdd <<<(N+255)/256, 256>>>();	// N hilos ejecutan el kernel en paralelo
  
  // comprobamos si la suma en DC ha sido correcta
  checkAdd <<<(N+255)/256, 256>>>();
    
  return 0;
} 
