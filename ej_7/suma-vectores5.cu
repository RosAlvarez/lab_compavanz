
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define N 65535*512+1000
#define numThreads 512
#define numBlocks 65535

__device__ int DA[N];
__device__  int DB[N];
__device__ int DC[N];
__device__ int stopKernel = 0;

__global__ void VecAdd(int elPerThread)
{
    for (int i = blockIdx.x*blockDim.x + threadIdx.x*elPerThread; i<elPerThread && i<N; i+=1)
      DC[i] = DA[i] + DB[i];
}

__global__ void initVec(int elPerThread)
{
      for (int i = blockIdx.x*blockDim.x + threadIdx.x*elPerThread; i<elPerThread && i<N; i+=1)
      {  
        DA[i] = -i;
        DB[i] = 3*i;
      }
}

__global__ void checkAdd(int elPerThread)
{
    if(stopKernel == 0)
    {
      for (int i = blockIdx.x*blockDim.x + threadIdx.x*elPerThread; i<elPerThread && i<N; i+=1)
      {  
        if (DC[i]!= (DA[i]+DB[i]))
        {
          printf("error en componente %d\n", i);
          stopKernel = 1;
        }
      }
    }
}

int main()
{ 
  int elPerThread  = round(N/(numBlocks*numThreads));

  hipError_t err = hipMalloc((void **)&elPerThread, sizeof(int));
  if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));

  // iniciamos los valores de los vectores DA y DB
  initVec <<<numBlocks, numThreads>>>(elPerThread);

  // llamamos al kernel
  VecAdd <<<numBlocks, numThreads>>>(elPerThread);	// N hilos ejecutan el kernel en paralelo
  
  // comprobamos si la suma en DC ha sido correcta
  checkAdd <<<numBlocks, numThreads>>>(elPerThread);
    
  return 0;
} 
